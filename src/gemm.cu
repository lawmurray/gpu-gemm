#include "hip/hip_runtime.h"
/*
 * Copyright 2024 Lawrence Murray.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>

/**
 * Matrix tile in global memory.
 * 
 * @tparam R Number of rows.
 * @tparam C Number of columns.
 * @tparam L Stride between columns.
 */
template<int R, int C, int L = R>
requires (R%4 == 0 && L >= R)
union global_tile {
  /**
   * Constructor.
   */
  __device__ global_tile(float* x) : x(x) {
    //
  }

  /**
   * Constructor.
   */
  template<int R1, int C1, int L1>
  __device__ global_tile(const global_tile<R1,C1,L1>& o, const int i,
      const int j) :
      x(&o.x[i + j*L1]) {
    //
  }

  float* __restrict__ x;
  float4* __restrict__ x4;
};

/**
 * Matrix tile in shared memory.
 * 
 * @tparam R Number of rows.
 * @tparam C Number of columns.
 * @tparam L Stride between columns.
 */
template<int R, int C, int L = R>
requires (R%4 == 0 && L >= R)
union shared_tile {
  /**
   * Constructor.
   */
  __device__ shared_tile(float* x) : x(x) {
    //
  }

  /**
   * Constructor.
   */
  template<int R1, int C1, int L1>
  __device__ shared_tile(const shared_tile<R1,C1,L1>& o, const int i,
      const int j) :
      x(&o.x[i + j*L1]) {
    //
  }

  /**
   * Copy into this tile from global memory using 32-bit loads.
   * 
   * @tparam T Number of threads in the group sharing the copy.
   * 
   * @param o Global memory tile.
   * @param i0 Row offset in @p o.
   * @param j0 Column offset in @p o.
   * @param t_id Id of this thread within the group sharing the copy.
   */
  template<int T, int R1, int C1, int L1>
  requires (T%R == 0)
  __device__ void copy(const global_tile<R1,C1,L1>& o, const int i0,
      const int j0, const int t_id) {
    int dst0 = __cvta_generic_to_shared(x);
    int i = t_id%R;
    int j1 = t_id/R;
    for (int s = 0; s < R*C/T; ++s) {
      int j = j1 + s*(T/R);
      int dst = dst0 + (i + j*L)*sizeof(float);
      const float* src = &o.x[i0 + i + (j0 + j)*L1];
      asm("cp.async.ca.shared.global [%0], [%1], %2;" :: "r"(dst), "l"(src),
          "n"(sizeof(float)));
    }
  }

  /**
   * Copy into this tile from global memory using 128-bit loads.
   * 
   * @tparam T Number of threads in the group sharing the copy.
   * 
   * @param o Global memory tile.
   * @param i0 Row offset in @p o.
   * @param j0 Column offset in @p o.
   * @param t_id Id of this thread within the group sharing the copy.
   */
  template<int T, int R1, int C1, int L1>
  requires (R%4 == 0 && L%4 == 0 && L1%4 == 0) && (T%(R/4) == 0)
  __device__ void copy4(const global_tile<R1,C1,L1>& o, const int i0,
      const int j0, const int t_id) {
    int dst0 = __cvta_generic_to_shared(x4);
    int i = t_id%(R/4);
    int j1 = t_id/(R/4);
    for (int s = 0; s < R*C/4/T; ++s) {
      int j = j1 + s*(T/(R/4));
      int dst = dst0 + (i + j*(L/4))*sizeof(float4);
      const float4* src = &o.x4[i0 + i + (j0 + j)*(L1/4)];
      asm("cp.async.cg.shared.global [%0], [%1], %2;" :: "r"(dst), "l"(src),
          "n"(sizeof(float4)));
    }
  }

  /**
   * Copy into this tile from global memory using 32-bit loads, with
   * transpose.
   * 
   * @tparam T Number of threads participating in the copy.
   * 
   * @param o Global memory tile.
   * @param i0 Row offset in @p o.
   * @param j0 Column offset in @p o.
   * @param t_id Thread id within the group.
   */
  template<int T, int R1, int C1, int L1>
  requires (T%C == 0)
  __device__ void copy_transpose(const global_tile<R1,C1,L1>& o, const int i0,
      const int j0, const int t_id) {
    int dst0 = __cvta_generic_to_shared(x);
    int i = t_id%C;
    int j1 = t_id/C;
    for (int s = 0; s < C*R/T; ++s) {
      int j = j1 + s*(T/C);
      int dst = dst0 + (j + i*L)*sizeof(float);
      const float* src = &o.x[i0 + i + (j0 + j)*L1];
      asm("cp.async.ca.shared.global.L2::128B [%0], [%1], %2;" :: "r"(dst),
          "l"(src), "n"(sizeof(float)));
    }
  }

  float* __restrict__ x;
  float4* __restrict__ x4;
};

/**
 * Matrix tile in registers.
 * 
 * @tparam R Number of rows.
 * @tparam C Number of columns.
 */
template<int R, int C>
union register_tile {
  /**
   * Load from a shared memory tile.
   */
  template<int R1, int C1, int L1>
  __device__ void load(const shared_tile<R1,C1,L1>& o, const int i0,
      const int j0) {
    for (int j = 0; j < C; ++j) {
      for (int i = 0; i < R; ++i) {
        x[i + j*R] = o.x[i0 + i + (j0 + j)*L1];
      }
    }
  }

  /**
   * Load from a shared memory tile.
   */
  template<int R1, int C1, int L1>
  requires (R%4 == 0 && L1%4 == 0)
  __device__ void load4(const shared_tile<R1,C1,L1>& o, const int i0,
      const int j0) {
    for (int j = 0; j < C; ++j) {
      for (int i = 0; i < R/4; ++i) {
        x4[i + j*(R/4)] = o.x4[i0 + i + (j0 + j)*(L1/4)];
      }
    }
  }

  /**
   * Store to a global memory tile.
   */
  template<int R1, int C1, int L1>
  __device__ void store(global_tile<R1,C1,L1>& o, const int i0,
      const int j0) {
    for (int j = 0; j < C; ++j) {
      for (int i = 0; i < R; ++i) {
        o.x[i0 + i + (j0 + j)*L1] = x[i + j*R];
      }
    }
  }

  /**
   * Store to a global memory tile.
   */
  template<int R1, int C1, int L1>
  requires (R%4 == 0 && L1%4 == 0)
  __device__ void store4(global_tile<R1,C1,L1>& o, const int i0,
      const int j0) {
    for (int j = 0; j < C; ++j) {
      for (int i = 0; i < R/4; ++i) {
        /* when storing, write through so as not to evict useful data from
          * inputs from the L2 cache */
        __stwt(&o.x4[i0 + i + (j0 + j)*(L1/4)], x4[i + j*(R/4)]);
      }
    }
  }

  /**
   * Multiply and add.
   * 
   * @param A First argument.
   * @param B Second argument.
   * 
   * Computes $AB$ and adds to this tile.
   */
  template<int K>
  __device__ void mad(const register_tile<R,K>& A,
      const register_tile<K,C>& B) {
    for (int k = 0; k < K; ++k) {
      for (int j = 0; j < C; ++j) {
        for (int i = 0; i < R; ++i) {
          x[i + j*R] += A.x[i + k*R]*B.x[k + j*K];
        }
      }
    }
  }

  /**
   * Multiply and add, with transpose of second argument.
   * 
   * @param A First argument.
   * @param B Second argument.
   * 
   * Computes $AB^\top$ and adds to this tile.
   */
  template<int K>
  __device__ void mad_transpose(const register_tile<R,K>& A,
      const register_tile<C,K>& B) {
    for (int k = 0; k < K; ++k) {
      for (int j = 0; j < C; ++j) {
        for (int i = 0; i < R; ++i) {
          x[i + j*R] += A.x[i + k*R]*B.x[j + k*C];
        }
      }
    }
  }

  float x[R*C]{0};
  float4 x4[R*C/4];
};

/**
 * Two-dimensional point.
 */
struct point2 { int i, j; };

/**
 * Coordinates for a given serial index along a two-dimensional Hilbert curve.
 * 
 * @tparam M Number of rows.
 * @tparam N Number of columns.
 * 
 * @param s Serial index.
 * 
 * @return Coordinates.
 */
template<int M, int N>
requires (M == N || M == N/2)
__device__ point2 hilbert2(const int s) {
  int i = 0, j = 0;
  int t = s;
  for (int k = 1; k < max(M, N); k *= 2) {
    int bi = 1 & (t/2);  // local gray code, u shape top left to bottom left
    int bj = 1 & (t ^ bi);
    if (bj == 0) {
      if (bi == 1) {
        i = k - 1 - i;  // flip up-down
        j = k - 1 - j;  // flip left-right
      }
      int tmp = i;  // transpose
      i = j;
      j = tmp;
    }
    i += k*bi;
    j += k*bj;
    t /= 4;
  }
  return {i, j};
}

/**
 * Difference between two matrices.
 * 
 * @tparam M Number of rows.
 * @tparam N Number of columns.
 * 
 * @param C Matrix.
 * @param D Matrix.
 * 
 * @return Maximum absolute element-wise difference.
 */
template<int M, int N>
float diff(const float* C, const float* D) {
  float mx = 0.0;
  for (int j = 0; j < N; ++j) {
    for (int i = 0; i < M; ++i) {
      mx = std::max(mx, std::abs(C[i + j*M] - D[i + j*M]));
    }
  }
  return mx;
}

/**
 * Matrix-matrix multiplication kernel.
 * 
 * @tparam M Number of rows of $A$ and $C$.
 * @tparam N Number of columns of $B$ and $C$.
 * @tparam K Number of columns of $A$ and rows of $B$.
 * 
 * @param A Matrix $A$.
 * @param B Matrix $B$.
 * @param C Matrix $C$.
 * 
 * Computes $C = AB$.
 */
template<int M, int N, int K>
__global__ void gemm_kernel(float* __restrict__ A, float* __restrict__ B,
    float* __restrict__ C) {
  /* config */
  constexpr int nthreads = 256;      // number of threads per block
  constexpr int wsize = 32;          // number of threads per warp
  constexpr int nstages = 4;  // number of asynchronous pipeline stages

  /* level 0 tile size (original matrices in global memory) */
  constexpr int M0 = M;
  constexpr int N0 = N;
  constexpr int K0 = K;

  /* level 1 tile size (global memory, thread block level) */
  constexpr int M1 = 256;
  constexpr int N1 = 128;
  constexpr int K1 = K0;  // must be K0

  /* level 2 tile size (shared memory, thread block level) */
  constexpr int M2 = 256;
  constexpr int N2 = 128;
  constexpr int K2 = 8;

  /* level 3 tile size (shared memory, warp level) */
  constexpr int M3 = 64;
  constexpr int N3 = 64;
  constexpr int K3 = K2;  // must be K2

  /* level 3 warp grid size */
  constexpr int M3_warps = M2/M3;
  constexpr int N3_warps = N2/N3;

  /* level 4 tile size (registers, thread level) */
  constexpr int M4 = 4;
  constexpr int N4 = 16;
  constexpr int K4 = 1;

  /* level 4 thread grid size */
  constexpr int M4_threads = 8;
  constexpr int N4_threads = 4;

  /* thread and warp ids */
  const int b_id = blockIdx.x;       // id of this block within the grid
  const int w_id = threadIdx.x/32;   // id of this warp within its block
  const int t_id = threadIdx.x%32;   // id of this thread within its warp
  const int row_id = w_id%M3_warps;  // id of row handled warp at level 3
  const int col_id = w_id/M3_warps;  // id of column handled warp at level 3

  /* barrier ids associated with the row and column handled by the warp */
  const int row_barrier = row_id;
  const int col_barrier = M3_warps + col_id;

  /* level 0 tiles (original matrices) */
  global_tile<M0,K0> A0(A);
  global_tile<K0,N0> B0(B);
  global_tile<M0,N0> C0(C);

  /* level 1 tiles */
  auto [b_i, b_j] = hilbert2<M0/M1,N0/N1>(b_id);
  global_tile<M1,K1,M0> A1(A0, b_i*M1, 0);
  global_tile<K1,N1,K0> B1(B0, 0, b_j*N1);
  global_tile<M1,N1,M0> C1(C0, b_i*M1, b_j*N1);

  /* level 3 buffers (B is transposed) */
  __shared__ float A_shared[M3_warps][nstages][M3*K3];
  __shared__ float BT_shared[N3_warps][nstages][N3*K3];

  /* level 4 tiles */
  register_tile<M4,N4> C4[N3/N4/N4_threads][M3/M4/M4_threads];

  /* multiply */
  const int r_id = t_id + row_id*wsize;
  const int c_id = t_id + col_id*wsize;
  for (int stage = 0; stage < nstages - 2; ++stage) {
    /* inlining the level 2 tiles here improves performance, possibly because
     * the loop is unrolled and may save 64-bit pointer operations; see below
     * for situation where inlining does not improve performance */
    shared_tile<N3,K3> BT3(BT_shared[col_id][stage]);
    BT3.copy_transpose<nthreads/N3_warps>(B1, stage*K2, col_id*N3, r_id);

    shared_tile<M3,K3> A3(A_shared[row_id][stage]);
    A3.copy4<nthreads/M3_warps>(A1, row_id*(M3/4), stage*K2, c_id);

    asm("cp.async.commit_group;");
  }

  const int B_offset = t_id/M4_threads*N4;
  const int A_offset = t_id%M4_threads*M4;

  for (int k2 = 0; k2 < K1/K2; ++k2) {
    int next_k = (k2 + (nstages - 2))%(K1/K2);
    int next_stage = (k2 + (nstages - 2))%nstages;

    /* inlining the level 2 tiles here *does not* improve performance,
     * possibly because the loop on k2 is not unrolled */
    global_tile<K2,N2,K0> next_B2(B1, next_k*K2, col_id*N3);
    shared_tile<N3,K3> next_BT3(BT_shared[col_id][next_stage]);
    next_BT3.copy_transpose<nthreads/N3_warps>(next_B2, 0, 0, r_id);

    global_tile<M2,K2,M0> next_A2(A1, row_id*M3, next_k*K2);
    shared_tile<M3,K3> next_A3(A_shared[row_id][next_stage]);
    next_A3.copy4<nthreads/M3_warps>(next_A2, 0, 0, c_id);

    asm("cp.async.commit_group;");

    asm("cp.async.wait_group %0;" :: "n"(nstages - 3));
    asm("barrier.sync.aligned %0, %1;" :: "r"(col_barrier), "n"(nthreads/N3_warps));
    asm("barrier.sync.aligned %0, %1;" :: "r"(row_barrier), "n"(nthreads/M3_warps));

    shared_tile<N3,K3> BT3(BT_shared[col_id][k2%nstages] + B_offset);
    shared_tile<M3,K3> A3(A_shared[row_id][k2%nstages] + A_offset);
    for (int k4 = 0; k4 < K3/K4; ++k4) {
      for (int j4 = 0; j4 < N3/N4/N4_threads; ++j4) {
        register_tile<N4,K4> BT4;
        BT4.load4(BT3, j4*(N4_threads*N4/4), k4*K4);
        for (int i4 = 0; i4 < M3/M4/M4_threads; ++i4) {
          register_tile<M4,K4> A4;
          A4.load4(A3, i4*(M4_threads*M4/4), k4*K4);
          C4[j4][i4].mad_transpose(A4, BT4);
        }
      }
    }
  }

  /* write final result */
  global_tile<M3,N3,M0> C3(C1, row_id*M3 + A_offset, col_id*N3 + B_offset);
  for (int j4 = 0; j4 < N3/N4/N4_threads; ++j4) {
    for (int i4 = 0; i4 < M3/M4/M4_threads; ++i4) {
      C4[j4][i4].store4(C3, i4*(M4_threads*M4/4), j4*(N4_threads*N4));
    }
  }
}

/**
 * Matrix-matrix multiplication.
 * 
 * @tparam M Number of rows of $A$ and $C$.
 * @tparam N Number of columns of $B$ and $C$.
 * @tparam K Number of columns of $A$ and rows of $B$.
 * 
 * @param A Matrix $A$.
 * @param B Matrix $B$.
 * @param C Matrix $C$.
 * 
 * Computes $C = AB$.
 */
template<int M, int N, int K>
void gemm(float* __restrict__ A, float* __restrict__ B,
    float* __restrict__ C) {
  dim3 block(256);
  dim3 grid((M/256)*(N/128));
  gemm_kernel<M,N,K><<<grid,block>>>(A, B, C);
}

/**
 * Cache flush kernel.
 * 
 * @tparam F Flush size.
 * 
 * @param f Vector.
 */
template<int F>
__global__ void flush_kernel(float* f) {
  f[threadIdx.x + blockIdx.x*blockDim.x] += 1.0f;
}

/**
 * Cache flush.
 * 
 * @tparam F Flush size.
 * 
 * @param f Vector.
 */
template<int F>
void flush(float* f) {
  dim3 block(256);
  dim3 grid(F/256);
  flush_kernel<F><<<grid,block>>>(f);
}

int main(int argc, char** argv) {
  /* initialize cublas */
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  /* initialize hiprand */
  constexpr int seed = 1;
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);

  auto run_test = [&]<int M, int N, int K, int ntrials, int nwarmup>() {
    /* number of floating point operations for TFLOPS numbers; each output
     * has M*N elements, each computed from K multiplications and K - 1
     * additions */
    constexpr long flop = M*N*(2l*K - 1l);

    /* cache flush size */
    constexpr int F = 32*1024*1024;

    /* initialize matrices; the output matrices, C and D, are allocated with
     * managed memory to support problem sizes somewhat beyond the available
     * device memory, while ensuring that the input matrices, A and B, are
     * always on device, which is more important for performance */
    float *A, *B, *C, *D, *f;
    hipMalloc((void**)&A, M*K*sizeof(float));
    hipMalloc((void**)&B, K*N*sizeof(float));
    hipMallocManaged((void**)&C, M*N*sizeof(float));
    hipMallocManaged((void**)&D, M*N*sizeof(float));
    hipMalloc((void**)&f, F*sizeof(float));
    hiprandGenerateUniform(gen, A, M*K);
    hiprandGenerateUniform(gen, B, K*N);
    hiprandGenerateUniform(gen, f, F);

    /* initialize events */
    hipEvent_t start1[ntrials], stop1[ntrials];
    hipEvent_t start2[ntrials], stop2[ntrials];
    for (int trial = 0; trial < ntrials; ++trial) {
      hipEventCreate(&start1[trial]);
      hipEventCreate(&stop1[trial]);
      hipEventCreate(&start2[trial]);
      hipEventCreate(&stop2[trial]);
    }

    /* initialize scalars */
    float scalar0 = 0.0f, scalar1 = 1.0f;

    /* warm up */
    for (int trial = 0; trial < nwarmup; ++trial) {
      hipMemPrefetchAsync(C, M*N*sizeof(float), 0);
      hiprandGenerateUniform(gen, C, M*N);  // clear output
      flush<F>(f);  // flush L2 cache
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &scalar1, A, M,
          B, K, &scalar0, C, M);

      hipMemPrefetchAsync(D, M*N*sizeof(float), 0);
      hiprandGenerateUniform(gen, D, M*N);  // clear output
      flush<F>(f);  // flush L2 cache
      gemm<M,N,K>(A, B, D);
    }

    /* benchmark */
    for (int trial = 0; trial < ntrials; ++trial) {
      hipMemPrefetchAsync(C, M*N*sizeof(float), 0);
      hiprandGenerateUniform(gen, C, M*N);  // clear output
      flush<F>(f);  // flush L2 cache
      hipEventRecord(start1[trial]);
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &scalar1, A, M,
          B, K, &scalar0, C, M);
      hipEventRecord(stop1[trial]);

      hipMemPrefetchAsync(D, M*N*sizeof(float), 0);
      hiprandGenerateUniform(gen, D, M*N);  // clear output
      flush<F>(f);  // flush L2 cache
      hipEventRecord(start2[trial]);
      gemm<M,N,K>(A, B, D);
      hipEventRecord(stop2[trial]);
    }

    /* results */
    float ms, ms1 = 0.0f, ms2 = 0.0f;
    for (int trial = 0; trial < ntrials; ++trial) {
      hipEventSynchronize(stop1[trial]);
      hipEventElapsedTime(&ms, start1[trial], stop1[trial]);
      ms1 += ms;

      hipEventSynchronize(stop2[trial]);
      hipEventElapsedTime(&ms, start2[trial], stop2[trial]);
      ms2 += ms;
    }
    ms1 /= ntrials;
    ms2 /= ntrials;
    float tflops1 = flop/ms1/1.0e9f;
    float tflops2 = flop/ms2/1.0e9f;
    float error = diff<M,M>(C, D);    

    /* report results */
    std::printf("| %6d | %6d | %6d | %11.3f | %11.3f | %15.3f | %15.3f | %6d | %9.3f |\n",
        M, N, K, ms1, ms2, tflops1, tflops2, ntrials, error);

    /* destroy events */
    for (int trial = 0; trial < ntrials; ++trial) {
      hipEventDestroy(start1[trial]);
      hipEventDestroy(stop1[trial]);
      hipEventDestroy(start2[trial]);
      hipEventDestroy(stop2[trial]);
    }

    /* free memory */
    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipFree(D);
  };

  /* table header */
  std::printf("|      M |      N |      K | cublas (ms) | custom (ms) | cublas (tflops) | custom (tflops) | trials |       err |\n");
  std::printf("| -----: | -----: | -----: | ----------: | ----------: | --------------: | --------------: | -----: | :-------: |\n");

  /* run tests and report */
  run_test.template operator()<2048,2048,2048,100,10>();
  run_test.template operator()<4096,4096,4096,100,10>();
  run_test.template operator()<8192,8192,8192,100,10>();
  run_test.template operator()<16384,16384,16384,100,10>();

  return 0;
}
